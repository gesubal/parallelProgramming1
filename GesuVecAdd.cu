//CS-4370 Parallel Programming for many core GPUs
//Name: Gesu Bal
/*
this is a simple cuda program calculating vector add for 2 dimensions on GPU device
I added two two-dimensional matrices A, B on the device GPU. 
After the device matrix addition kernel function is invoked, and the addition result is transferred back to the CPU. 
The program will also compute the  sum matrix of matrices A and B using the CPU.  
Then the program compares the device-computed result with the CPU-computed result. 
If it matches, it prints out Test PASSED to the screen before exiting.
*/
#include<stdio.h>
#include<hip/hip_runtime.h>
int N,blocksize;

//gpu function for addition
__global__ void add_gpu(int *d_a, int *d_b, int *d_c, int N)
{
     
     int row=blockIdx.y*blockDim.y+threadIdx.y;
     int col=blockIdx.x*blockDim.x+threadIdx.x;
     //int index =i+(j*N); 
     if((row <N) && (col <N))
     {
       
           d_c[row*N+col]=d_a[row*N+col]+d_b[row*N+col];
	   
      }
      
      
}

//cpu function for addition
void add_matrix_cpu(int *a, int *b, int *cpu_c, int N)
{ 
int i, j; 
for (i=0;i<N;i++) { 
       for (j=0;j<N;j++) {
             
             cpu_c[i*N+j]=a[i*N+j]+b[i*N+j];
     } 
  } 
} 

//match cpu and gpu results
int verify(int * a, int * b, int N)
{   
    int i,j;
    int error=0;
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
		    if(a[i*N+j]!=b[i*N+j])
		    {
		     error++; 
		    }
		}
	}
	
	if(error==0)
	{
	  printf("CPU and GPU results matched: Test Passed \n");
	}
	else
	{
	  printf("CPU and GPU results did not match");
	}
    return 1;
    
}

//print matrix fucntion
int printMatrix(int *a,int N)
{
  int i,j;
  for (i=0;i<N;i++)
    {
        for (j=0;j<N;j++)
        {
          printf("%d\t",a[i*N+j]);
        }
        printf("\n");
    }
return 1;
  
}

int main()
{
    //user input    
    int r, col;
	printf("Select one of the following options for vector addition: \n");
	printf("Press a for matrix size 8 * 8 \n");
	printf("Press b for matrix size 64 * 64 \n");
	printf("Press c for matrix size 128 * 128 \n");
	printf("Press d for matrix size 500 * 500 \n");
	printf("Press e for matrix size 1000 * 1000 \n");
    printf("Press any other key for exiting \n");
	char ch;
	scanf("%c",&ch);
	switch(ch)
        {
            case 'a':
                r=8;
		col=8;
		N=8;
		blocksize=4;
		printf("Array size is 8 * 8 \n");
		
                break;
            case 'b':
                r=64;
		col=64;
		N=64;
		blocksize=16;
		printf("Array size is 64 * 64 \n");
		
                break;
            case 'c':
                r=128;
		col=128;
		N=128;
		blocksize=16;
		printf("Array size is 128 * 128 \n");
		
                break;
	    case 'd':
                r=500;
		col=500;
		N=500;
		blocksize=16;
		printf("Array size is 500 * 500 \n");
		
                break;
            case 'e':
                r=1000;
		col=1000;
		N=1000;
		blocksize=16;
		printf("Array size is 1000 * 1000 \n");
		
                break;
	    default:
		exit(1);
                break;            
	}
  
    //vector initialization
	int *a, *b, *c, *cpu_c, *d_a, *d_b, *d_c;
	
	int a_size=r*col;
	int b_size=r*col;
	int c_size=r*col;
	int cpu_c_size=r*col;
	
    
    //memory allocation for vectors on host	
	a=(int*)malloc(sizeof(int)*a_size);
	b=(int*)malloc(sizeof(int)*b_size);
	c=(int*)malloc(sizeof(int)*c_size);
	cpu_c=(int*)malloc(sizeof(int)*cpu_c_size);
		
	
	//matrix initialization
    int i,j;
	int init=1325;
        for (i=0;i<N;i++)
	{
		for (j=0;j<N;j++)
		{
		    init=3125*init%65536;
		    a[i*col+j]=((init-32768)/16384);
		    b[i*col+j]=(init%1000);
		}
	}

	int cudaret=hipMalloc((void **)(&d_a),(N*N)*sizeof(int));
	if(cudaret!=hipSuccess)
	{printf("memory was not allocated on device \n");}
	
	hipMalloc((void **)(&d_b),(N*N)*sizeof(int));
	hipMalloc((void **)(&d_c),(N*N)*sizeof(int));
	

	//copying contents of a and b to device arrays
	hipMemcpy(d_a,a,(N*N)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,(N*N)*sizeof(int),hipMemcpyHostToDevice);
	
	//Initializing block count and block size
	dim3 dimBlock(blocksize,blocksize,1); 
	int blockCount_x = (N - 1)/(double(blocksize))+1;//Get number of blocks needed per direction.
	int blockCount_y = (N - 1)/(double(blocksize))+1;
	printf("the number of the thread blocks in x direction will be %d\n", blockCount_x);
	printf("the number of the thread blocks in y direction will be %d\n", blockCount_y);
	dim3 dimGrid(blockCount_x,blockCount_y,1);
        
	//calling CPU program
	printf("calculating results for CPU vector addition \n");
	printf("---------\n");
	add_matrix_cpu(a,b,cpu_c,N);
    
    //printMatrix(a,N);
    //pritnMatrix(b,N);
    //printMatrix(cpu_c,N);
	
	//call kernel for gpu functioning
	printf("calling kernel for gpu computations for vector addition \n");
	printf("---------\n");
	add_gpu<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,N);
	printf("calculating results for gpu \n");
	printf("---------\n");
	
    //copying resulting back to cpu from gpu
    hipMemcpy(c,d_c,(N*N)*sizeof(int),hipMemcpyDeviceToHost);
        
	//matching cpu and gpu results
	printf("comparing results for CPU and GPU computations \n");
	printf("---------\n");
	verify(c,cpu_c,N);
	//printMatrix(c,N);
	
    //Deallocating memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
	
    return 0;
}
